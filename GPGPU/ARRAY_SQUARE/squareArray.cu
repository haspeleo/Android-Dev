// * http://llpanorama.wordpress.com/2008/05/21/my-first-cuda-program/
/*square array on GPU Written by me */

#include<stdio.h>
#include<hip/hip_runtime.h>

#include<time.h>

/******************************************************************/
//kernel that execute on the CUDA device "Tesla" for example 
/******************************************************************/

__global__ void square_array(float *a, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx<N) a[idx]= a[idx] * a[idx];
        }

/******************************************************************/
//main  function that executes the host : the CPU 
/******************************************************************/

int main(void) {

	//for time mesurement 
	clock_t start, end;
	double elapsed;
	start = clock();

  float *a_h; //I point  the host array 
  float *a_d; //I point  the device array

  const int N=10000; //Number of the elements in the array 
  size_t size = N * sizeof (float);

  a_h = (float *)malloc(size); //I allocate the array on the host(allocation on CPU) 
  hipMalloc((void **)&a_d, size);//I allocate the array on the device (allocation on GPU)

  for (int i=0;i<N;i++)
  a_h[i] = (float)i;//loop for initializing the array on the CPU

  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);//copy array elements from CPU to GPU

 //do Calulation on GPU
 int block_size = 4;
 int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);

//Call to kernel function
 square_array <<< n_blocks, block_size >>> (a_d, N);

 //Bring back results to host array 
 hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);

 //Print the result array 
 for (int i=0;i< N;i++)
 printf("%d %f\n ", i, a_h[i] );

//Free host and device arrays
 free(a_h);
 hipFree(a_d);

//get time
end = clock();
elapsed = ((double) end - start) / CLOCKS_PER_SEC;
printf("%.2f secondes entre start et end. \n", elapsed);

}	


#include "hip/hip_runtime.h"
/* compile :  nvcc -o vectorSum gpuVectorSum.cu  */

#include <stdio.h>

#define N 10000

/******************************************************************/
//kernel that execute on the CUDA device "Tesla" for example 
/******************************************************************/
__global__ void add (int *a, int *b, int *c) {

        int tid = blockIdx.x;    // handle the data at this index
        if (tid < N)
            c[tid] = a[tid] + b[tid];

} 
/******************************************************************/
//main  function that executes the host : the CPU 
/******************************************************************/

int main () {
	
	clock_t start, end;
	double elapsed;
	start = clock();

        int a[N], b[N], c[N];
        int *a_d, *b_d, *c_d;

        //Allocate the arrays on the device
        hipMalloc((void**)&a_d, N * sizeof(int));
        hipMalloc((void**)&b_d, N * sizeof(int));
        hipMalloc((void**)&c_d, N * sizeof(int));


        //init A, B arrays
        for (int i = 0; i < N; i++) {
                a[i] = -i;
                b[i] = i * i;
        }

        //copy Array A and B to device 
          hipMemcpy(a_d, a, N * sizeof(int), hipMemcpyHostToDevice);//copy array elements from CPU to GPU
          hipMemcpy(b_d, b, N * sizeof(int), hipMemcpyHostToDevice);//copy array elements from CPU to GPU

        //Call kernel function to be executed on GPU
        //add<<<n_blocks, block_size>>>(a_d, b_d, c_d);
        add<<<N, 64>>>(a_d, b_d, c_d);

         //Bring back results to host array 
 	hipMemcpy(c, c_d, N * sizeof(int), hipMemcpyDeviceToHost);
	
 	//Print the result array 
	for (int i=0; i<N; i++) {
	    printf( "%d + %d = %d\n", a[i], b[i], c[i] );
	}	
	
	end = clock();
	elapsed = ((double)end - start) / CLOCKS_PER_SEC;
	
	printf("ELAPSED TIME = %.4f\n", elapsed);

	//free the device array	
	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

return 0;
}

